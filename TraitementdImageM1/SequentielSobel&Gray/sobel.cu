#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>
#include <iostream>

__global__ void grayscale( unsigned char * rgb, unsigned char * g, std::size_t cols, std::size_t rows ) {
  auto i = blockIdx.x*blockDim.x+ threadIdx.x;
  auto j = blockIdx.y*blockDim.y+ threadIdx.y;
  if( i < cols && j < rows ) {
    g[ j * cols + i ] = (
			 307 * rgb[ 3 * ( j * cols + i ) ]
		       + 604 * rgb[ 3 * ( j * cols + i ) + 1 ]
		       + 113 * rgb[  3 * ( j * cols + i ) + 2 ]
		       ) / 1024;
  }
}

__global__ void sobel(unsigned char * g_in, unsigned char * g, std::size_t cols, std::size_t rows  ){
  auto i = blockIdx.x*blockDim.x+ threadIdx.x;
  auto j = blockIdx.y*blockDim.y+ threadIdx.y;
  int h, v, res;


  if(i>0 && j>0 && i < cols-1 && j < rows-1){
    // Horizontal
  	h =     g_in[((j - 1) * cols + i - 1) * 1 ] -     g_in[((j - 1) * cols + i + 1) * 1 ]
  	  + 2 * g_in[( j      * cols + i - 1) * 1 ] - 2 * g_in[( j      * cols + i + 1) * 1 ]
  	  +     g_in[((j + 1) * cols + i - 1) * 1 ] -     g_in[((j + 1) * cols + i + 1) * 1 ];

  	// Vertical

  	v =     g_in[((j - 1) * cols + i - 1) * 1 ] -     g_in[((j + 1) * cols + i - 1) * 1 ]
  	  + 2 * g_in[((j - 1) * cols + i    ) * 1 ] - 2 * g_in[((j + 1) * cols + i    ) * 1 ]
  	  +     g_in[((j - 1) * cols + i + 1) * 1 ] -     g_in[((j + 1) * cols + i + 1) * 1 ];

  	//h = h > 255 ? 255 : h;
  	//v = v > 255 ? 255 : v;
  	res = h*h + v*v;
  	res = res > 255*255 ? res = 255*255 : res;
  	g[(j * cols + i) ] = sqrt( (float) res);
  }
}

int main()
{
  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );

  auto rgb = m_in.data;

  auto rows = m_in.rows;
  auto cols = m_in.cols;

  std::vector< unsigned char > g( rows * cols ); // image de sortie grayscale.
  std::vector< unsigned char > gsobel( rows * cols ); // image de sortie sobel.

  cv::Mat m_out( rows, cols, CV_8UC1, gsobel.data() );

  unsigned char * rgb_d;
  unsigned char * g_d;
  unsigned char * gsobel_d;
  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  hipMalloc(&rgb_d,3*rows*cols); // allocation pour l'image d'entrée sur le device.
  hipMalloc(&g_d,rows*cols); // allocation pour l'image de sortie du grayscale sur le device.

  hipMemcpy(rgb_d,rgb,3*rows*cols,hipMemcpyHostToDevice); // copie de l'image d'entrée vers le device.

  dim3 t( 32, 32 );
  dim3 b( ( cols - 1) / t.x + 1 , ( rows - 1 ) / t.y + 1 );
  grayscale<<< b, t >>>( rgb_d,g_d,cols,rows );

  //hipMemcpy(gsobel.data(),g_d,rows*cols, hipMemcpyDeviceToHost); // récupération de l'image en niveaux de gris sur l'hôte.

  hipDeviceSynchronize();
  auto err = hipGetLastError();
  if( err != hipSuccess )
  {
    std::cerr << hipGetErrorString( err ) << std::endl;
  }

  hipMalloc(&gsobel_d, rows*cols);
  sobel<<< b, t >>>(g_d,gsobel_d,cols,rows);

  hipDeviceSynchronize();
  err = hipGetLastError();
  if( err != hipSuccess )
  {
    std::cerr << " " << hipGetErrorString( err ) << std::endl;
  }

  hipMemcpy(gsobel.data(),gsobel_d,rows*cols, hipMemcpyDeviceToHost);


  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float elapseTime;
  hipEventElapsedTime(&elapseTime,start,stop);
  std::cout<<elapseTime<<"ms"<<std::endl;

  cv::imwrite( "out.jpg", m_out ); // sauvegarde de l'image.

  hipFree( rgb_d );
  hipFree( g_d);
  hipFree( gsobel_d);

  return 0;
}

#include "hip/hip_runtime.h"

#include <opencv2/opencv.hpp>
#include <vector>
#include <math.h>

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#include<chrono>
#include <IL/il.h>

__global__ void sobel(unsigned char * g, std::size_t cols, std::size_t rows ) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;
int h, v, res;

 if( i < cols && j < rows ) {
		
	// Horizontal
	h =     g[((j - 1) * cols + i - 1) ] -   g[((j - 1) * cols + i + 1)]
	  + 2 * g[( j      * cols + i - 1) ] - 2 * g[( j      * cols + i + 1)]
	  +     g[((j + 1) * cols + i - 1)] -     g[((j + 1) * cols + i + 1)];

	// Vertical

	v =     g[((j - 1) * cols + i - 1) ] -     g[((j + 1) * cols + i - 1) ]
	  + 2 * g[((j - 1) * cols + i    )] - 2 * g[((j + 1) * cols + i    )]
	  +     g[((j - 1) * cols + i + 1)] -     g[((j + 1) * cols + i + 1)];

	//h = h > 255 ? 255 : h;
	//v = v > 255 ? 255 : v;

	res = h*h + v*v;
	res = res > 255*255 ? res = 255*255 : res;

	g[(j * cols + i)] = sqrtf(res);

      }

    
}



int main()
{
  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );
	
  auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;
  std::vector< unsigned char > g( rows * cols );
  cv::Mat m_out( rows, cols, CV_8UC1, g.data() );
  unsigned char * rgb_d;
  unsigned char * g_d;
  hipError_t err;
 
  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );
  hipEventRecord( start );

  err=hipMalloc( &rgb_d, 3 * rows * cols );
   	if( err != hipSuccess){
       std::cerr << hipGetErrorString(err)<<"Erreur lors de l'allocation"<< std::endl;;
   }
  err=hipMalloc( &g_d, rows * cols );
	if( err != hipSuccess){
       std::cerr << hipGetErrorString(err)<<"Erreur lors de l'allocation"<< std::endl;;
   }
  err=hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );
	if( err != hipSuccess){
       std::cerr << hipGetErrorString(err)<<"Erreur lors de la copie host to device"<< std::endl;;
   }
  dim3 t( 32, 32 );
  dim3 b( ( cols - 1) / t.x + 1 , ( rows - 1 ) / t.y + 1 );

  hipDeviceSynchronize();
  sobel<<< b, t >>>( rgb_d,g_d, cols, rows );
// Récupération du code erreur du kernel en cas de plantage.
  hipDeviceSynchronize(); // Attente de la fin d'exécution du kernel.
  hipError_t err = hipGetLastError();
  if( err != hipSuccess )
  {
    std::cerr << hipGetErrorString( err ); // récupération du message associé au code erreur.
  }


  err=hipMemcpy( g.data(), g_d, rows * cols, hipMemcpyDeviceToHost );
	if( err != hipSuccess){
       std::cerr << hipGetErrorString(err)<<"Erreur lors de la copie devise to host"<< std::endl;;
   }
  hipEventRecord( stop );

  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );
std::cout << "Total Temps mis pour kernel sobel : " << duration << "ms\n";
 
  cv::imwrite( "out.jpg", m_out );
  hipFree( rgb_d);
  hipFree( g_d);
  return 0;
}

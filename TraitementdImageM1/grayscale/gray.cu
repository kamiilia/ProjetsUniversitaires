#include "hip/hip_runtime.h"


#include <opencv2/opencv.hpp>
#include <vector>

__global__ void grayscale( unsigned char * rgb, unsigned char * g, std::size_t cols, std::size_t rows ) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;
  if( i < cols && j < rows ) {
    g[ j * cols + i ] = (
			 307 * rgb[ 3 * ( j * cols + i ) ]
			 + 604 * rgb[ 3 * ( j * cols + i ) + 1 ]
			 + 113 * rgb[  3 * ( j * cols + i ) + 2 ]
			 ) / 1024;
  }
}


int main()
{
  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;
  std::vector< unsigned char > g( rows * cols );
  cv::Mat m_out( rows, cols, CV_8UC1, g.data() );
  unsigned char * rgb_d;
  unsigned char * g_d;

  hipError_t err;
 
  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );
  hipEventRecord( start );

  err= hipMalloc( &rgb_d, 3 * rows * cols );
  if( err != hipSuccess){
       std::cerr << hipGetErrorString(err)<<"Erreur lors de l'allocation"<< std::endl;;
   }
  err=hipMalloc( &g_d, rows * cols );
     if( err != hipSuccess){
       std::cerr << hipGetErrorString(err)<<"Erreur lors de l'allocation"<< std::endl;;
   }
  err=hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );
if( err != hipSuccess){
       std::cerr << hipGetErrorString(err)<<"Erreur lors de la copie host to device"<< std::endl;;
   }
  dim3 t( 32, 32 );
  dim3 b( ( cols - 1) / t.x + 1 , ( rows - 1 ) / t.y + 1 );
  grayscale<<< b, t >>>( rgb_d, g_d, cols, rows );
 // Récupération du code erreur du kernel en cas de plantage.
  hipDeviceSynchronize(); // Attente de la fin d'exécution du kernel.
  hipError_t err = hipGetLastError();
  if( err != hipSuccess )
  {
    std::cerr << hipGetErrorString( err ); // récupération du message associé au code erreur.
  }

  err=hipMemcpy( g.data(), g_d, rows * cols, hipMemcpyDeviceToHost );
  if( err != hipSuccess){
       std::cerr << hipGetErrorString(err)<<"Erreur lors de la copie devise to host"<< std::endl;;
   }
  hipEventRecord( stop );

  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );

  std::cout << "Total Temps mis pour gray kernel est : " << duration << "ms\n";
  cv::imwrite( "out.jpg", m_out );
  hipFree( rgb_d);
  hipFree( g_d);
  return 0;
}

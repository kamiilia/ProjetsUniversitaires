#include "hip/hip_runtime.h"

__global__ void fusion(unsigned char * rgb, unsigned char * g, unsigned char * g2, std::size_t cols, std::size_t rows ) { // image d'entrée en couleur et image de sortie en couleur

  auto i=blockIdx.x*blockDim.x + threadIdx.x;
  auto j= blockIdx.y*blockDim.y + threadIdx.y;
  int h, v, res;
  if( i > 0 && i < (cols-1) && j > 0 && j < (rows-1) ) {
     
               //debut grayscale
              
               g[((j - 1) * cols + i - 1) ] = (307 * rgb [ 3 * ((j - 1) * cols + i - 1)] + 604 * rgb[ 3 * ((j - 1) * cols + i - 1) + 1 ] + 113 * rgb[  3 * ((j - 1) * cols + i - 1) + 2 ]) / 1024;

                g[((j - 1) * cols + i + 1) ] = (307 * rgb [ 3 * ((j - 1) * cols + i + 1)] + 604 * rgb[ 3 * ((j - 1) * cols + i + 1) + 1 ] + 113 * rgb[  3 * ((j - 1) * cols + i + 1) + 2 ]) / 1024;
               
                g[( j      * cols + i - 1) ] = (307 * rgb [ 3 * ( j      * cols + i - 1)] + 604 * rgb[ 3 * ( j      * cols + i - 1) + 1 ] + 113 * rgb[ 3 * ( j      * cols + i - 1) + 2 ]) / 1024;


                g[( j      * cols + i + 1)] = (307 * rgb [ 3 * ( j      * cols + i + 1)] + 604 * rgb[ 3 * ( j      * cols + i + 1) + 1 ] + 113 * rgb[ 3 * ( j      * cols + i + 1) + 2 ]) / 1024;

                g[((j + 1) * cols + i - 1) ] = (307 * rgb [ 3 * ((j + 1) * cols + i - 1)] + 604 * rgb[ 3 * ((j + 1) * cols + i - 1) + 1 ] + 113 * rgb[ 3 * ((j + 1) * cols + i - 1) + 2 ]) / 1024;

                g[((j + 1) * cols + i + 1)] = (307 * rgb [ 3 * ((j + 1) * cols + i + 1)] + 604 * rgb[ 3 * ((j + 1) * cols + i + 1) + 1 ] + 113 * rgb[ 3 * ((j + 1) * cols + i + 1) + 2 ]) / 1024;
               
                //fin grayscale
               
                // debut sobel

                // Horizontal
                h =     g[((j - 1) * cols + i - 1)] -     g[((j - 1) * cols + i + 1) ]
                  + 2 * g[( j      * cols + i - 1)] - 2 * g[( j      * cols + i + 1) ]
                  +     g[((j + 1) * cols + i - 1) ] -     g[((j + 1) * cols + i + 1) ];

                // Vertical

                v =     g[((j - 1) * cols + i - 1) ] -     g[((j + 1) * cols + i - 1) ]
                  + 2 * g[((j - 1) * cols + i    ) ] - 2 * g[((j + 1) * cols + i    ) ]
                  +     g[((j - 1) * cols + i + 1) ] -     g[((j + 1) * cols + i + 1) ];

                res = h*h + v*v;
                res = res > 255*255 ? res = 255*255 : res;

                g2[ (j * cols + i)] = sqrtf(res);
                //fin sobel
               
      }
    
  
  
 
}






int main()
{
 
  //Debut sobel et grayscale
 
  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );

  auto rgb = m_in.data;  //l'image d'entrée en couleur

  auto rows = m_in.rows;
  auto cols = m_in.cols;

  std::vector< unsigned char > g( rows * cols ); // image de sortie en niveau de gris

  cv::Mat m_out( rows, cols, CV_8UC1, g.data() );

  unsigned char * rgb_d;
  unsigned char * g_d;
  unsigned char * g_d3;
 
  hipError_t err;
 
  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );

  hipEventRecord( start );

  err=hipMalloc( &rgb_d,3*rows*cols ); // allocation pour l'image d'entrée sur le device. 3 = rgb
   if( err != hipSuccess){
       std::cerr << hipGetErrorString(err)<<"Erreur lors de l'allocation"<< std::endl;;
   }
  err=hipMalloc( &g_d,rows*cols  ); // allocation pour l'image de sortie sur le device.
   if( err != hipSuccess){
       std::cerr << hipGetErrorString(err)<<"Erreur lors de l'allocation"<< std::endl;;
   }
  err=hipMalloc( &g_d3,rows*cols ); // allocation pour l'image de sortie sur le device.
   if( err != hipSuccess){
       std::cerr << hipGetErrorString(err)<<"Erreur lors de l'allocation"<< std::endl;;
   }

  err=hipMemcpy(rgb_d,rgb, 3*rows*cols ,hipMemcpyHostToDevice ); // copie de l'image d'entrée vers le device. la destination en premier
   if( err != hipSuccess){
       std::cerr << hipGetErrorString(err)<<"Erreur lors de la copie host to device"<< std::endl;;
   }

  dim3 t( 32, 32 );
  dim3 b( ( cols - 1) / t.x + 1 , ( rows - 1 ) / t.y + 1 );
 
 
 
 
  fusion<<< b, t >>>( rgb_d, g_d,g_d3, cols, rows );
 
  // Récupération du code erreur du kernel en cas de plantage
  hipDeviceSynchronize();
  err= hipGetLastError();
  if( err!= hipSuccess){
      std::cerr << hipGetErrorString(err) <<"Erreur lors de l'appel du kernel fusion"<<std::endl;;
  }
 
  //Récupération du code erreur pour les fonctions CUDA synchrones

   err = hipMemcpy( g.data(), g_d3, rows*cols, hipMemcpyDeviceToHost); // récupération de l'image en niveaux de gris sur l'hôte.
   if( err != hipSuccess){
       std::cerr << hipGetErrorString(err)<<"Erreur lors de la copie devise to host"<< std::endl;;
   }


  hipDeviceSynchronize();
  hipEventRecord( stop );
  hipEventSynchronize( stop );

  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );

  std::cout << "Total Temps mis pour la fusion des deux kernels : " << duration << "ms\n";
  cv::imwrite( "out_fusion.jpg", m_out ); // sauvegarde de l'image.

  hipFree( rgb_d );
  hipFree( g_d);
  hipFree( g_d3);
  //Fin grayscale
 
 

  return 0;
}

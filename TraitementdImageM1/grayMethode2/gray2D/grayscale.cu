#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
extern"C"{
#include <ppm.h>
}

// kernel CUDA
__global__ void gray(pixel* img_in, pixel* img_out, int cols, int rows){
  // récupération des indices globaux dans la grille 2D pour les
  // dimensions X et Y
  int y = blockIdx.y*blockDim.y + threadIdx.y;
  int x = blockIdx.x*blockDim.x + threadIdx.x;

  // on s'assure de ne pas sortir des limites de l'image
  if (x < cols && y < rows){
    int idx = y*cols+x;
    pixel pix = img_in[idx];
    int gray = 
      0.21f * pix.r + 0.71f * pix.g + 0.07f * pix.b;

    img_out[idx].r = gray;
    img_out[idx].g = gray;
    img_out[idx].b = gray;
    
  }
  
}

// Lecture d'un fichier ppm
pixel * readppm (const char* fname, int* cols, int* rows, pixval* maxval, int* format){
  FILE* img_in;
  
  img_in = fopen(fname, "r");
  ppm_readppminit(img_in, cols, rows, maxval, format);

  pixel* out = (pixel*)malloc ((*cols)*(*rows)*sizeof(pixel));
  int i;
  for (i =0; i < *rows; i++){
    ppm_readppmrow(img_in, out+(i*(*cols)), *cols, *maxval, *format);
  }
  return out;
  
}

// Ecriture d'un fichier ppm 
void writeppm(const char* fname, pixel* out, int cols, int rows, pixval maxval, int format){
  FILE* img_out;
  img_out = fopen(fname, "w+");
  int i;
  
  ppm_writeppminit(img_out, cols, rows, maxval, format);


  for (i =0; i < rows; i++){
    ppm_writeppmrow(img_out, out+(i*(cols)), cols, maxval, 1);
  }
}


int main(){
  
  pixel *ppm_in, *ppm_out = NULL;
  pixel *d_ppm_in, *d_ppm_out = NULL;

  int cols, rows;
  pixval maxval;
  int format;
  int err;
  
  ppm_in = readppm("lena.ppm", &cols, &rows, &maxval, &format);
  
  long size = cols*rows*sizeof(pixel);  

  ppm_out = (pixel*)malloc(size);
  hipMalloc(&d_ppm_in, size);
  hipMalloc(&d_ppm_out, size);
  
  
  err = hipMemcpy(d_ppm_in, ppm_in, size, hipMemcpyHostToDevice);
  if (err != 0) printf("Error %d\n", err);
  
  dim3 DimBlock(16, 16,1);
  dim3 DimGrid((rows + DimBlock.x -1)/DimBlock.x, (cols + DimBlock.y -1)/DimBlock.y, 1);
  
  gray<<<DimGrid, DimBlock>>>(d_ppm_in, d_ppm_out, cols, rows);
  
  err = hipMemcpy(ppm_out, d_ppm_out, size, hipMemcpyDeviceToHost);
  if (err != 0) {
    printf("Error %d\n", err);
    return err;
  }
  
  writeppm("lena_gray.ppm", ppm_out, cols, rows, maxval, 1);
  
  hipFree(d_ppm_in);
  hipFree(d_ppm_out);
  free(ppm_in);
  free(ppm_out);
  return 0;
}

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
extern"C"{
#include <pgm.h>
#include <ppm.h>
}

__global__ void gray_oneD(pixel *in, pixel *out, int cols, int rows){
	//1D
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	out[i].b = out[i].g = out[i].r = (in[i].b + in[i].g + in[i].r) / 3;
}


pixel * readppm (const char* fname, int* cols, int* rows, pixval* maxval, int* format){
  FILE* img_in;
  img_in = fopen(fname, "r");
  
  ppm_readppminit(img_in, cols, rows, maxval, format);
  
  pixel* out = (pixel*)malloc ((*cols)*(*rows)*sizeof(pixel));
  int i;
  for (i =0; i < *rows; i++){
    ppm_readppmrow(img_in, out+(i*(*cols)), *cols, *maxval, *format);
  }
  return out;
}

void writeppm(const char* fname, pixel* out, int cols, int rows, pixval maxval, int format){
  FILE* img_out;
  img_out = fopen(fname, "w+");
  int i;
  
  ppm_writeppminit(img_out, cols, rows, maxval, format);


  for (i =0; i < rows; i++){
    ppm_writeppmrow(img_out, out+(i*(cols)), cols, maxval, 1);
  }
}




int main(){
  
  const char* fname = "peppers.ppm";
  const char* fname_out = "peppers_out.ppm";
  
  pixel *ppm_in, *ppm_out; // en mémoire CPU
  pixel *d_ppm_in, *d_ppm_out; // en mémoire GPU
  
  int cols, rows, format;
  pixval maxval;
 
  //Lire image dans ppm_in
	ppm_in = readppm(fname, &cols, &rows, &maxval, &format);

  //Allocation mémoire 
	ppm_out = (pixel*)malloc(cols*rows*sizeof(pixel));
	hipMalloc(&d_ppm_in, cols*rows*sizeof(pixel));
	hipMalloc(&d_ppm_out, cols*rows*sizeof(pixel));
  
  //Copie de ppm_in dans d_ppm_in (en mémoire GPU)
  hipMemcpy(d_ppm_in,ppm_in, cols*rows*sizeof(pixel), hipMemcpyHostToDevice);
	
  
  //Déclaration tailles de grilles et blocs + lancement kernel
  int blockSize = rows;
  int gridSize = cols;

  gray_oneD<<<blockSize, gridSize>>>(d_ppm_in, d_ppm_out, cols, rows);
  
  
  //Copie du résultat dans ppm_out
    hipMemcpy(ppm_out,d_ppm_out, cols*rows*sizeof(pixel), hipMemcpyDeviceToHost);
 
  //Ecriture du fichier ppm
	writeppm(fname_out, ppm_out, cols, rows, maxval, format);
  
  //Libération mémoire
  hipFree(d_ppm_in);
  hipFree(d_ppm_out);
  
  free(ppm_out);
  free(ppm_in);
  return 0;
}



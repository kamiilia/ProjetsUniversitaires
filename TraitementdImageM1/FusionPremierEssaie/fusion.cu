#include "hip/hip_runtime.h"

#include <opencv2/opencv.hpp>
#include <vector>
#include <math.h>

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#include<chrono>
#include <IL/il.h>

__global__ void fusion(unsigned char * rgb,unsigned char * g, std::size_t cols, std::size_t rows ) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;
int h, v, res;

	// Debut grayscale
if(i < cols && j < rows ) {
    g[ j * cols + i ] = (
			 307 * rgb[ 3 * ( j * cols + i ) ]
			 + 604 * rgb[ 3 * ( j * cols + i ) + 1 ]
			 + 113 * rgb[  3 * ( j * cols + i ) + 2 ]
			 ) / 1024;
  }

	// Fin grayscale

	// Debut sobel
 if( i>0 && j>0 && i<(cols-1) && j<(rows-1)) {

	// Horizontal
	h =     g[((j - 1) * cols + i - 1)] -   g[((j - 1) * cols + i + 1)]
	  + 2 * g[( j      * cols + i - 1)] - 2 * g[( j      * cols + i + 1)]
	  +     g[((j + 1) * cols + i - 1)] -     g[((j + 1) * cols + i + 1)];

	// Vertical

	v =     g[((j - 1) * cols + i - 1)] -     g[((j + 1) * cols + i - 1)]
	  + 2 * g[((j - 1) * cols + i    )] - 2 * g[((j + 1) * cols + i    )]
	  +     g[((j - 1) * cols + i + 1)] -     g[((j + 1) * cols + i + 1)];


	res = h*h + v*v;
	res = res > 255*255 ? res = 255*255 : res;

	g[(j * cols + i)] = sqrtf(res);
	// Fin sobel

      }



int main()
{
  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );
	
  auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;
  std::vector< unsigned char > g( rows * cols );
  cv::Mat m_out( rows, cols, CV_8UC1, g.data() );
  unsigned char * rgb_d;
  unsigned char * g_d;

  hipMalloc( &rgb_d, 3 * rows * cols );
  hipMalloc( &g_d, rows * cols );
  hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );
  dim3 t( 32, 32 );
  dim3 b( ( cols - 1) / t.x + 1 , ( rows - 1 ) / t.y + 1 );
  
  fusion<<< b, t >>>( rgb_d,g_d, cols, rows );


  hipMemcpy( g.data(), g_d, rows * cols, hipMemcpyDeviceToHost );

  cv::imwrite( "out.jpg", m_out );
  hipFree( rgb_d);
  hipFree( g_d);
  return 0;
}

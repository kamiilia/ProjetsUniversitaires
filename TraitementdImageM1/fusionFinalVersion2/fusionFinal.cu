#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>

__global__ void fusion(unsigned char * rgb, unsigned char *out, unsigned char *out2, std::size_t cols , std::size_t rows ) {
  auto i = blockIdx.x * blockDim.x + threadIdx.x;
  auto j = blockIdx.y * blockDim.y + threadIdx.y;
  
  i-= blockIdx.x*2;
  j-= blockIdx.y*2; 
  
	if( i < cols && j < rows ) {
   out[ j * cols + i ] = (
			 307 * rgb[ 3 * ( j * cols + i ) ]
			 + 604 * rgb[ 3 * ( j * cols + i ) + 1 ]
			 + 113 * rgb[  3 * ( j * cols + i ) + 2 ]
			 ) / 1024;
  }
    
  __syncthreads();
    
  int h, v, res;
  if( i < cols-1 && j < rows-1 && i>0 && j>0) {
    // Horizontal
			h =    out[((j - 1) * cols + i - 1)] -    out[((j - 1) * cols + i + 1)]
				+ 2 *out[( j      * cols + i - 1)] - 2 *out[( j      * cols + i + 1)]
				+    out[((j + 1) * cols + i - 1)] -    out[((j + 1) * cols + i + 1)];
			// Vertical
			v =    out[((j - 1) * cols + i - 1)] -    out[((j + 1) * cols + i - 1)]
				+ 2 *out[((j - 1) * cols + i    )] - 2 *out[((j + 1) * cols + i    )]
				+    out[((j - 1) * cols + i + 1)] -    out[((j + 1) * cols + i + 1)];
			res = h*h + v*v;
			res = res > 255*255 ? res = 255*255 : res;
			out2[(j * cols + i)] = sqrt((float)res);
  }

}


int main()
{
  cv::Mat m_in = cv::imread("in.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;
  std::vector< unsigned char >out( rows * cols );
  cv::Mat m_out( rows, cols, CV_8UC1,out.data() );
  hipError_t err = hipSuccess;
  
  unsigned char * rgb_d;
  unsigned char *out_d;
  unsigned char *out2_d;
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  
  err = hipMalloc( &rgb_d, 3 * rows * cols );
  if(err!=hipSuccess){
    std::cerr << hipGetErrorString(err)<< std :: endl;
    return 1;
  }
  err = hipMalloc( &out_d, rows * cols );
  if(err!=hipSuccess){
    std::cerr << hipGetErrorString(err)<< std :: endl;
    return 1;
  }
  err = hipMalloc( &out2_d, rows * cols );
  if(err!=hipSuccess){
    std::cerr << hipGetErrorString(err)<< std :: endl;
    return 1;
  }
  
  err = hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );
  if(err!=hipSuccess){
    std::cerr << hipGetErrorString(err)<< std :: endl;
    return 1;
  }
  
  dim3 t( 32, 32 );
  dim3 b( ( cols - 1) / (t.x - 2) + 1 , ( rows - 1 ) / (t.y - 2) + 1 );
  
  fusion<<<b,t>>>(rgb_d,out_d,out2_d, cols,rows);

  err=hipGetLastError();
  if(err!=hipSuccess){
    std::cerr << "error grayscaleSobel : " << hipGetErrorString(err)<< std :: endl;
    return 1;
  }
  
  err = hipMemcpy(outdata(),out2_d, rows * cols, hipMemcpyDeviceToHost);
  if(err!=hipSuccess){
    std::cerr << "error hipMemcpy : "<< hipGetErrorString(err)<< std :: endl;
    return 1;
  }
  
  hipEventRecord(stop);
	hipEventSynchronize(stop);
	float elapseTime;
	hipEventElapsedTime(&elapseTime,start,stop);
	std::cout<< elapseTime << " ms" << std::endl;
  
  cv::imwrite( "out.jpg", m_out );
  hipFree(rgb_d);
  hipFree(out_d);
  hipFree(out2_d );
  return 0;
}
